#include "hip/hip_runtime.h"
#include "tensor.h"

#ifdef __HIPCC__
#include <hip/hip_runtime.h>

__global__ void elementwiseSumKernel(float *result, const float *a, const float *b, size_t size)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        result[idx] = a[idx] + b[idx];
    }
}

void Tensor::elementwiseSumGPU(float *result, const float *a, const float *b, size_t size) const
{
    constexpr size_t blockSize = 256;
    const size_t numBlocks     = (size + blockSize - 1) / blockSize;

    elementwiseSumKernel<<<numBlocks, blockSize>>>(result, a, b, size);

    hipDeviceSynchronize();
}

#endif
