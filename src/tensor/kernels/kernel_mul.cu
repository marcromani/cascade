#include "hip/hip_runtime.h"
#include "kernel_mul.h"
#include "tensor.h"
#include "tensor_data.h"

#include <cstddef>
#include <hip/hip_runtime.h>
#include <vector>

namespace cascade
{
__global__ void kernelMulForward_(float *result, const float *x, const float *y, size_t size)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        result[idx] = x[idx] * y[idx];
    }
}

void kernelMulForward(const Tensor &result, const Tensor &x, const Tensor &y)
{
    constexpr size_t blockSize = 256;

    size_t size = result.size();

    size_t numBlocks = (size + blockSize - 1) / blockSize;

    kernelMulForward_<<<numBlocks, blockSize>>>(
        result.data_->deviceData.get(), x.data_->deviceData.get(), y.data_->deviceData.get(), size);

    hipDeviceSynchronize();
}

__global__ void kernelMulBackward_(float *dx,
                                   float *dy,
                                   const float *x,
                                   const float *y,
                                   size_t size,
                                   const size_t *shape,
                                   size_t dims)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        size_t indices[1024];  // Big enough number of dimensions to avoid hipMalloc

        size_t stride = 1;

        for (int i = dims - 1; i >= 0; --i)
        {
            indices[i] = (idx / stride) % shape[i];
            stride *= shape[i];
        }

        bool allEqual = true;

        for (size_t i = 0; (i < dims / 2) && allEqual; ++i)
        {
            allEqual = allEqual && (indices[i] == indices[i + dims / 2]);
        }

        if (allEqual)
        {
            size_t childIdx = 0;
            stride          = 1;

            for (int i = dims / 2 - 1; i >= 0; --i)
            {
                childIdx += indices[i] * stride;
                stride *= shape[i];
            }

            dx[idx] = y[childIdx];
            dy[idx] = x[childIdx];
        }
        else
        {
            dx[idx] = 0.f;
            dy[idx] = 0.f;
        }
    }
}

void kernelMulBackward(const Tensor &x, const Tensor &y)
{
    size_t dims = x.shape().size();

    size_t *shapePtr;
    hipMalloc(reinterpret_cast<void **>(&shapePtr), 2 * dims * sizeof(size_t));
    hipMemcpy(shapePtr, x.shape().data(), dims * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(shapePtr + dims, x.shape().data(), dims * sizeof(size_t), hipMemcpyHostToDevice);

    constexpr size_t blockSize = 256;

    size_t size = x.size();

    size_t numBlocks = (size * size + blockSize - 1) / blockSize;

    kernelMulBackward_<<<numBlocks, blockSize>>>(x.data_->deviceGrad.get(),
                                                 y.data_->deviceGrad.get(),
                                                 x.data_->deviceData.get(),
                                                 y.data_->deviceData.get(),
                                                 size * size,
                                                 shapePtr,
                                                 2 * dims);

    hipDeviceSynchronize();

    hipFree(shapePtr);
}
}  // namespace cascade
